#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define ROW_SIZE 32
#define COL_SIZE 32

__global__ void matAdd_2D_index(float* _dA, float* _dB, float* _dC)
{
	unsigned int col = threadIdx.x;
	unsigned int row = threadIdx.y;
	unsigned int index = row * blockDim.x + col;

	_dC[index] = _dA[index] + _dB[index];
}

int main(void)
{
	// set the matrix on host
	float A[ROW_SIZE][COL_SIZE] = { 0 };
	float B[ROW_SIZE][COL_SIZE] = { 0 };
	float C[ROW_SIZE][COL_SIZE] = { 0 };
	float hC[ROW_SIZE][COL_SIZE] = { 0 };

	// set memory space on the device memory
	int matSize = ROW_SIZE * COL_SIZE;
	float* dA = NULL;
	float* dB = NULL;
	float* dC = NULL;
	hipMalloc(&dA, sizeof(float) * matSize); hipMemset(dA, 0, sizeof(float) * matSize);
	hipMalloc(&dB, sizeof(float) * matSize); hipMemset(dB, 0, sizeof(float) * matSize);
	hipMalloc(&dC, sizeof(float) * matSize); hipMemset(dC, 0, sizeof(float) * matSize);

	// generting input matrix
	for (int iRow = 0; iRow < ROW_SIZE; iRow++)
		for (int iCol = 0; iCol < COL_SIZE; iCol++) {
			A[iRow][iCol] = rand() % 100;
			B[iRow][iCol] = rand() % 100;
			C[iRow][iCol] = A[iRow][iCol] + B[iRow][iCol];
		}

	// Copy matrix A and B from host to device (dA, dB)
	hipMemcpy(dA, A, sizeof(float) * matSize, hipMemcpyHostToDevice);
	hipMemcpy(dB, B, sizeof(float) * matSize, hipMemcpyHostToDevice);

	// kernel call
	dim3 blockDim(COL_SIZE, ROW_SIZE);
	matAdd_2D_index << <1, blockDim >> > (dA, dB, dC);

	// Copy matrix C from device (dC) to host
	hipMemcpy(hC, dC, sizeof(float) * matSize, hipMemcpyDeviceToHost);

	// validation
	bool isCorrect = true;
	for (int iRow = 0; iRow < ROW_SIZE; iRow++)
		for (int iCol = 0; iCol < COL_SIZE; iCol++) {
			if (hC[iRow][iCol] != C[iRow][iCol]) {
				isCorrect = false;
				break;
			}
		}

	if (isCorrect) printf("GPU works well!\n");
	else printf("GPU fail to make correct resutl(s)..\n");

	return 0;
}